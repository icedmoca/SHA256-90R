/*********************************************************************
* GPU Batch Benchmark for SHA256-90R
* Tests scaling with 100k+ concurrent messages
*********************************************************************/

#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
#include <time.h>

// SHA256-90R constants
#define SHA256_90R_ROUNDS 90
#define BLOCK_SIZE 64
#define DIGEST_SIZE 32

// GPU configuration
#define THREADS_PER_BLOCK 256
#define MAX_BLOCKS 65535

// Device constants
__constant__ uint32_t d_k[96];

// SHA256-90R device functions
__device__ uint32_t rotr(uint32_t x, int n) {
    return (x >> n) | (x << (32 - n));
}

__device__ uint32_t ch(uint32_t x, uint32_t y, uint32_t z) {
    return (x & y) ^ (~x & z);
}

__device__ uint32_t maj(uint32_t x, uint32_t y, uint32_t z) {
    return (x & y) ^ (x & z) ^ (y & z);
}

__device__ uint32_t sig0(uint32_t x) {
    return rotr(x, 7) ^ rotr(x, 18) ^ (x >> 3);
}

__device__ uint32_t sig1(uint32_t x) {
    return rotr(x, 17) ^ rotr(x, 19) ^ (x >> 10);
}

__device__ uint32_t ep0(uint32_t x) {
    return rotr(x, 2) ^ rotr(x, 13) ^ rotr(x, 22);
}

__device__ uint32_t ep1(uint32_t x) {
    return rotr(x, 6) ^ rotr(x, 11) ^ rotr(x, 25);
}

// SHA256-90R kernel - processes one message per thread
__global__ void sha256_90r_batch_kernel(const uint8_t* messages, uint32_t* hashes, int num_messages) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_messages) return;
    
    // Initialize state
    uint32_t state[8] = {
        0x6a09e667, 0xbb67ae85, 0x3c6ef372, 0xa54ff53a,
        0x510e527f, 0x9b05688c, 0x1f83d9ab, 0x5be0cd19
    };
    
    // Load message (simplified - assumes 64-byte messages)
    uint32_t w[96];
    const uint8_t* msg = messages + idx * BLOCK_SIZE;
    
    // Load and byte swap
    #pragma unroll 16
    for (int i = 0; i < 16; i++) {
        w[i] = ((uint32_t)msg[i*4] << 24) | ((uint32_t)msg[i*4+1] << 16) |
               ((uint32_t)msg[i*4+2] << 8) | (uint32_t)msg[i*4+3];
    }
    
    // Message expansion
    #pragma unroll
    for (int i = 16; i < 90; i++) {
        w[i] = sig1(w[i-2]) + w[i-7] + sig0(w[i-15]) + w[i-16];
    }
    
    // Compression
    uint32_t a = state[0], b = state[1], c = state[2], d = state[3];
    uint32_t e = state[4], f = state[5], g = state[6], h = state[7];
    
    #pragma unroll
    for (int i = 0; i < 90; i++) {
        uint32_t t1 = h + ep1(e) + ch(e, f, g) + d_k[i] + w[i];
        uint32_t t2 = ep0(a) + maj(a, b, c);
        h = g; g = f; f = e; e = d + t1;
        d = c; c = b; b = a; a = t1 + t2;
    }
    
    // Store final state
    state[0] += a; state[1] += b; state[2] += c; state[3] += d;
    state[4] += e; state[5] += f; state[6] += g; state[7] += h;
    
    // Write output
    uint32_t* out = hashes + idx * 8;
    #pragma unroll 8
    for (int i = 0; i < 8; i++) {
        out[i] = state[i];
    }
}

// Host constants
const uint32_t k_host[96] = {
    0x428a2f98,0x71374491,0xb5c0fbcf,0xe9b5dba5,0x3956c25b,0x59f111f1,0x923f82a4,0xab1c5ed5,
    0xd807aa98,0x12835b01,0x243185be,0x550c7dc3,0x72be5d74,0x80deb1fe,0x9bdc06a7,0xc19bf174,
    0xe49b69c1,0xefbe4786,0x0fc19dc6,0x240ca1cc,0x2de92c6f,0x4a7484aa,0x5cb0a9dc,0x76f988da,
    0x983e5152,0xa831c66d,0xb00327c8,0xbf597fc7,0xc6e00bf3,0xd5a79147,0x06ca6351,0x14292967,
    0x27b70a85,0x2e1b2138,0x4d2c6dfc,0x53380d13,0x650a7354,0x766a0abb,0x81c2c92e,0x92722c85,
    0xa2bfe8a1,0xa81a664b,0xc24b8b70,0xc76c51a3,0xd192e819,0xd6990624,0xf40e3585,0x106aa070,
    0x19a4c116,0x1e376c08,0x2748774c,0x34b0bcb5,0x391c0cb3,0x4ed8aa4a,0x5b9cca4f,0x682e6ff3,
    0x748f82ee,0x78a5636f,0x84c87814,0x8cc70208,0x90befffa,0xa4506ceb,0xbef9a3f7,0xc67178f2,
    // Extended for rounds 64-89
    0x240ca1cc,0x2de92c6f,0x4a7484aa,0x5cb0a9dc,0x76f988da,0x983e5152,
    0xa831c66d,0xb00327c8,0xbf597fc7,0xc6e00bf3,0xd5a79147,0x06ca6351,
    0x14292967,0x27b70a85,0x2e1b2138,0x4d2c6dfc,0x53380d13,0x650a7354,
    0x766a0abb,0x81c2c92e,0x92722c85,0xa2bfe8a1,0xa81a664b,0xc24b8b70,
    0xc76c51a3,0xd192e819
};

double get_time() {
    struct timespec ts;
    clock_gettime(CLOCK_MONOTONIC, &ts);
    return ts.tv_sec + ts.tv_nsec * 1e-9;
}

int main() {
    printf("SHA256-90R GPU Batch Scaling Benchmark\n");
    printf("======================================\n\n");
    
    // Initialize CUDA
    int device_count;
    hipGetDeviceCount(&device_count);
    if (device_count == 0) {
        printf("No CUDA devices found!\n");
        return 1;
    }
    
    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, 0);
    printf("GPU: %s\n", props.name);
    printf("Compute Capability: %d.%d\n", props.major, props.minor);
    printf("Max threads/block: %d\n", props.maxThreadsPerBlock);
    printf("Max blocks: %d\n\n", props.maxGridSize[0]);
    
    // Copy constants to device
    hipMemcpyToSymbol(HIP_SYMBOL(d_k), k_host, sizeof(k_host));
    
    // Test different batch sizes
    int batch_sizes[] = {1000, 10000, 100000, 1000000, 10000000};
    int num_tests = sizeof(batch_sizes) / sizeof(batch_sizes[0]);
    
    printf("Batch Size | Time (s) | Throughput (Gbps) | Hashes/sec\n");
    printf("-----------|----------|-------------------|------------\n");
    
    for (int test = 0; test < num_tests; test++) {
        int num_messages = batch_sizes[test];
        size_t input_size = (size_t)num_messages * BLOCK_SIZE;
        size_t output_size = (size_t)num_messages * DIGEST_SIZE;
        
        // Allocate memory
        uint8_t* h_messages = (uint8_t*)malloc(input_size);
        uint32_t* h_hashes = (uint32_t*)malloc(output_size);
        
        if (!h_messages || !h_hashes) {
            printf("Failed to allocate host memory for %d messages\n", num_messages);
            continue;
        }
        
        // Initialize test data
        for (size_t i = 0; i < input_size; i++) {
            h_messages[i] = (uint8_t)(i & 0xFF);
        }
        
        // Allocate device memory
        uint8_t* d_messages;
        uint32_t* d_hashes;
        
        if (hipMalloc(&d_messages, input_size) != hipSuccess ||
            hipMalloc(&d_hashes, output_size) != hipSuccess) {
            printf("Failed to allocate device memory for %d messages\n", num_messages);
            free(h_messages);
            free(h_hashes);
            continue;
        }
        
        // Copy input to device
        hipMemcpy(d_messages, h_messages, input_size, hipMemcpyHostToDevice);
        
        // Configure kernel
        int blocks = (num_messages + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
        if (blocks > MAX_BLOCKS) blocks = MAX_BLOCKS;
        
        // Warmup
        sha256_90r_batch_kernel<<<blocks, THREADS_PER_BLOCK>>>(d_messages, d_hashes, num_messages);
        hipDeviceSynchronize();
        
        // Benchmark
        double start = get_time();
        
        sha256_90r_batch_kernel<<<blocks, THREADS_PER_BLOCK>>>(d_messages, d_hashes, num_messages);
        hipDeviceSynchronize();
        
        double end = get_time();
        double elapsed = end - start;
        
        // Calculate throughput
        double bytes_processed = (double)input_size;
        double throughput_gbps = (bytes_processed * 8) / (elapsed * 1e9);
        double hashes_per_sec = num_messages / elapsed;
        
        printf("%10d | %8.4f | %17.2f | %.2e\n", 
               num_messages, elapsed, throughput_gbps, hashes_per_sec);
        
        // Cleanup
        hipFree(d_messages);
        hipFree(d_hashes);
        free(h_messages);
        free(h_hashes);
    }
    
    printf("\nNote: GPU performance is limited by kernel launch overhead for small batches.\n");
    printf("      Optimal performance requires 100k+ concurrent messages.\n");
    
    return 0;
}
