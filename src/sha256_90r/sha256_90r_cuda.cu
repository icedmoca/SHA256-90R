#include "hip/hip_runtime.h"
/**
 * SHA256-90R CUDA Implementation
 * GPU-accelerated batch hashing for maximum throughput
 */

#include <hip/hip_runtime.h>
#include <>

// SHA-256 constants aligned for GPU memory
__constant__ uint32_t d_k_90r[96] = {
	0x428a2f98,0x71374491,0xb5c0fbcf,0xe9b5dba5,0x3956c25b,0x59f111f1,0x923f82a4,0xab1c5ed5,
	0xd807aa98,0x12835b01,0x243185be,0x550c7dc3,0x72be5d74,0x80deb1fe,0x9bdc06a7,0xc19bf174,
	0xe49b69c1,0xefbe4786,0x0fc19dc6,0x240ca1cc,0x2de92c6f,0x4a7484aa,0x5cb0a9dc,0x76f988da,
	0x983e5152,0xa831c66d,0xb00327c8,0xbf597fc7,0xc6e00bf3,0xd5a79147,0x06ca6351,0x14292967,
	0x27b70a85,0x2e1b2138,0x4d2c6dfc,0x53380d13,0x650a7354,0x766a0abb,0x81c2c92e,0x92722c85,
	0xa2bfe8a1,0xa81a664b,0xc24b8b70,0xc76c51a3,0xd192e819,0xd6990624,0xf40e3585,0x106aa070,
	0x19a4c116,0x1e376c08,0x2748774c,0x34b0bcb5,0x391c0cb3,0x4ed8aa4a,0x5b9cca4f,0x682e6ff3,
	0x748f82ee,0x78a5636f,0x84c87814,0x8cc70208,0x90befffa,0xa4506ceb,0xbef9a3f7,0xc67178f2,
	// Extended constants for SHA-256-90R
	0xc67178f2,0xca273ece,0xd186b8c7,0xeada7dd6,0xf57d4f7f,0x06f067aa,0x0a637dc5,0x113f9804,
	0x1b710b35,0x28db77f5,0x32caab7b,0x3c9ebe0a,0x431d67c4,0x4cc5d4be,0x597f299c,0x5fcb6fab,
	0x6c44198c,0x7ba0ea2d,0x7eabf2d0,0x8dbe8d03,0x90bb1721,0x99a2ad45,0x9f86e289,0xa84c4472,
	0xb3df34fc,0xb99bb8d7,
	// Padding for alignment
	0x00000000,0x00000000,0x00000000,0x00000000,0x00000000,0x00000000
};

// Device functions for SHA-256 operations
__device__ __forceinline__ uint32_t ROTR(uint32_t x, int n) {
	return (x >> n) | (x << (32 - n));
}

__device__ __forceinline__ uint32_t CH(uint32_t x, uint32_t y, uint32_t z) {
	return (x & y) ^ (~x & z);
}

__device__ __forceinline__ uint32_t MAJ(uint32_t x, uint32_t y, uint32_t z) {
	return (x & y) ^ (x & z) ^ (y & z);
}

__device__ __forceinline__ uint32_t EP0(uint32_t x) {
	return ROTR(x, 2) ^ ROTR(x, 13) ^ ROTR(x, 22);
}

__device__ __forceinline__ uint32_t EP1(uint32_t x) {
	return ROTR(x, 6) ^ ROTR(x, 11) ^ ROTR(x, 25);
}

__device__ __forceinline__ uint32_t SIG0(uint32_t x) {
	return ROTR(x, 7) ^ ROTR(x, 18) ^ (x >> 3);
}

__device__ __forceinline__ uint32_t SIG1(uint32_t x) {
	return ROTR(x, 17) ^ ROTR(x, 19) ^ (x >> 10);
}

// Macro-generated unrolled compression pipeline
#define SHA256_90R_GPU_ROUND(a,b,c,d,e,f,g,h,t1,t2,k_idx,m_idx) \
	t1 = h + EP1(e) + CH(e,f,g) + d_k_90r[k_idx] + m[m_idx]; \
	t2 = EP0(a) + MAJ(a,b,c); \
	h = g; g = f; f = e; e = d + t1; d = c; c = b; b = a; a = t1 + t2;

// Constant-time CUDA kernel for batch SHA256-90R processing
// Hardened against timing side-channels with uniform execution patterns
__global__ void sha256_90r_warp_kernel(
	const uint8_t* __restrict__ input_data,
	uint32_t* __restrict__ output_states,
	size_t num_blocks
) {
	const int global_tid = blockIdx.x * blockDim.x + threadIdx.x;
	const int warp_id = global_tid / 32;
	const int lane_id = threadIdx.x % 32;

	// Each warp processes one block
	for (size_t block_idx = warp_id; block_idx < num_blocks; block_idx += (gridDim.x * blockDim.x / 32)) {
		// Load input data for this block
		const uint8_t* data = input_data + block_idx * 64;

		// Shared memory for message expansion (90 words per warp)
		// Use fixed-size shared memory to ensure constant access patterns
		__shared__ uint32_t shared_m[90 * 32]; // 90 words * max warps per block
		uint32_t* m = shared_m + (threadIdx.x / 32) * 90;

		// Constant-time message expansion - all threads participate uniformly
		// Each thread handles exactly one word position, using arithmetic to select data
		for (int word_idx = 0; word_idx < 16; ++word_idx) {
			int byte_offset = word_idx * 4;
			uint32_t word = (data[byte_offset] << 24) | (data[byte_offset + 1] << 16) |
						   (data[byte_offset + 2] << 8) | data[byte_offset + 3);

			// Use arithmetic selection instead of conditional assignment
			uint32_t mask = (lane_id == word_idx) ? 0xFFFFFFFF : 0;
			m[word_idx] = (word & mask) | (m[word_idx] & ~mask);
		}
		__syncwarp();

		// Constant-time extended message expansion
		// All threads compute all words but only store when appropriate
		for (int i = 16; i < 90; ++i) {
			uint32_t m_i_minus_16 = m[i - 16];
			uint32_t m_i_minus_15 = m[i - 15];
			uint32_t m_i_minus_7 = m[i - 7];
			uint32_t m_i_minus_2 = m[i - 2];

			uint32_t new_word = SIG1(m_i_minus_2) + m_i_minus_7 + SIG0(m_i_minus_15) + m_i_minus_16;

			// Use arithmetic selection instead of conditional assignment
			uint32_t mask = (lane_id == (i % 32)) ? 0xFFFFFFFF : 0;
			m[i] = (new_word & mask) | (m[i] & ~mask);
		}
		__syncwarp();

		// Initialize state using arithmetic selection (constant-time)
		uint32_t initial_states[8] = {
			0x6a09e667, 0xbb67ae85, 0x3c6ef372, 0xa54ff53a,
			0x510e527f, 0x9b05688c, 0x1f83d9ab, 0x5be0cd19
		};

		uint32_t a = 0, b = 0, c = 0, d = 0, e = 0, f = 0, g = 0, h = 0;

		// Distribute initial state using arithmetic operations
		for (int state_idx = 0; state_idx < 8; ++state_idx) {
			uint32_t state_val = initial_states[state_idx];
			uint32_t mask = (lane_id == state_idx) ? 0xFFFFFFFF : 0;

			a = (state_val & mask) | (a & ~mask);
			b = (state_val & mask) | (b & ~mask);
			c = (state_val & mask) | (c & ~mask);
			d = (state_val & mask) | (d & ~mask);
			e = (state_val & mask) | (e & ~mask);
			f = (state_val & mask) | (f & ~mask);
			g = (state_val & mask) | (g & ~mask);
			h = (state_val & mask) | (h & ~mask);
		}

		uint32_t t1, t2;

		// Constant-time compression rounds
		// All threads execute the same operations every round
		for (int round = 0; round < 90; ++round) {
			// All threads compute the same message word selection
			uint32_t m_round = m[round];

			// All threads compute round operations identically
			t1 = h + EP1(e) + CH(e, f, g) + d_k_90r[round] + m_round;
			t2 = EP0(a) + MAJ(a, b, c);

			h = g;
			g = f;
			f = e;
			e = d + t1;
			d = c;
			c = b;
			b = a;
			a = t1 + t2;
		}

		// Constant-time state collection and storage
		// All threads participate in state collection
		uint32_t final_a = a, final_b = b, final_c = c, final_d = d;
		uint32_t final_e = e, final_f = f, final_g = g, final_h = h;

		// Store final state - all threads write, but only lane 0's data is valid
		if (lane_id < 8) {
			uint32_t* out_state = output_states + block_idx * 8;
			uint32_t state_values[8] = {final_a, final_b, final_c, final_d,
									   final_e, final_f, final_g, final_h};
			out_state[lane_id] = initial_states[lane_id] + state_values[lane_id];
		}
	}
}

#undef SHA256_90R_GPU_ROUND

// Host function to launch the warp-optimized CUDA kernel
extern "C" hipError_t launch_sha256_90r_cuda_batch(
	const uint8_t* input_data,
	uint32_t* output_states,
	size_t num_blocks,
	int threads_per_block = 256
) {
	hipError_t cudaStatus;

	// Allocate device memory
	uint8_t* d_input = nullptr;
	uint32_t* d_output = nullptr;

	size_t input_size = num_blocks * 64 * sizeof(uint8_t);
	size_t output_size = num_blocks * 8 * sizeof(uint32_t);

	cudaStatus = hipMalloc(&d_input, input_size);
	if (cudaStatus != hipSuccess) return cudaStatus;

	cudaStatus = hipMalloc(&d_output, output_size);
	if (cudaStatus != hipSuccess) {
		hipFree(d_input);
		return cudaStatus;
	}

	// Copy input data to device
	cudaStatus = hipMemcpy(d_input, input_data, input_size, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		hipFree(d_input);
		hipFree(d_output);
		return cudaStatus;
	}

	// Calculate grid dimensions for warp-level processing
	// Each warp (32 threads) processes one block
	int warps_per_block = threads_per_block / 32;
	int blocks_per_grid = (num_blocks + warps_per_block - 1) / warps_per_block;

	// Prefetch constants to GPU
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(d_k_90r), k_90r, sizeof(k_90r));
	if (cudaStatus != hipSuccess) {
		hipFree(d_input);
		hipFree(d_output);
		return cudaStatus;
	}

	// Launch warp-optimized kernel
	sha256_90r_warp_kernel<<<blocks_per_grid, threads_per_block>>>(
		d_input, d_output, num_blocks
	);

	// Check for kernel launch errors
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		hipFree(d_input);
		hipFree(d_output);
		return cudaStatus;
	}

	// Copy results back to host
	cudaStatus = hipMemcpy(output_states, d_output, output_size, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		hipFree(d_input);
		hipFree(d_output);
		return cudaStatus;
	}

	// Cleanup
	hipFree(d_input);
	hipFree(d_output);

	return hipSuccess;
}

// CUDA kernel for regression testing constant-time behavior
__global__ void sha256_90r_constant_time_test_kernel(
	const uint8_t* __restrict__ input_data1,
	const uint8_t* __restrict__ input_data2,
	uint32_t* __restrict__ output_states,
	size_t num_blocks,
	uint64_t* __restrict__ timing_results
) {
	const int global_tid = blockIdx.x * blockDim.x + threadIdx.x;
	const int warp_id = global_tid / 32;
	const int lane_id = threadIdx.x % 32;

	// Each warp processes pairs of blocks for timing comparison
	for (size_t block_idx = warp_id; block_idx < num_blocks; block_idx += (gridDim.x * blockDim.x / 32)) {
		uint64_t start_time = clock64();

		// Process first input (all zeros)
		const uint8_t* data1 = input_data1 + block_idx * 64;
		__shared__ uint32_t shared_m1[90 * 32];
		uint32_t* m1 = shared_m1 + (threadIdx.x / 32) * 90;

		// Constant-time message expansion for input 1
		for (int word_idx = 0; word_idx < 16; ++word_idx) {
			int byte_offset = word_idx * 4;
			uint32_t word = (data1[byte_offset] << 24) | (data1[byte_offset + 1] << 16) |
						   (data1[byte_offset + 2] << 8) | data1[byte_offset + 3);
			uint32_t mask = (lane_id == word_idx) ? 0xFFFFFFFF : 0;
			m1[word_idx] = (word & mask) | (m1[word_idx] & ~mask);
		}
		__syncwarp();

		for (int i = 16; i < 90; ++i) {
			uint32_t m_i_minus_16 = m1[i - 16];
			uint32_t m_i_minus_15 = m1[i - 15];
			uint32_t m_i_minus_7 = m1[i - 7];
			uint32_t m_i_minus_2 = m1[i - 2];
			uint32_t new_word = SIG1(m_i_minus_2) + m_i_minus_7 + SIG0(m_i_minus_15) + m_i_minus_16;
			uint32_t mask = (lane_id == (i % 32)) ? 0xFFFFFFFF : 0;
			m1[i] = (new_word & mask) | (m1[i] & ~mask);
		}
		__syncwarp();

		// Process first input through compression
		uint32_t initial_states[8] = {0x6a09e667, 0xbb67ae85, 0x3c6ef372, 0xa54ff53a,
									 0x510e527f, 0x9b05688c, 0x1f83d9ab, 0x5be0cd19};
		uint32_t a1 = 0, b1 = 0, c1 = 0, d1 = 0, e1 = 0, f1 = 0, g1 = 0, h1 = 0;

		for (int state_idx = 0; state_idx < 8; ++state_idx) {
			uint32_t state_val = initial_states[state_idx];
			uint32_t mask = (lane_id == state_idx) ? 0xFFFFFFFF : 0;
			a1 = (state_val & mask) | (a1 & ~mask);
			b1 = (state_val & mask) | (b1 & ~mask);
			c1 = (state_val & mask) | (c1 & ~mask);
			d1 = (state_val & mask) | (d1 & ~mask);
			e1 = (state_val & mask) | (e1 & ~mask);
			f1 = (state_val & mask) | (f1 & ~mask);
			g1 = (state_val & mask) | (g1 & ~mask);
			h1 = (state_val & mask) | (h1 & ~mask);
		}

		for (int round = 0; round < 90; ++round) {
			uint32_t m_round = m1[round];
			uint32_t t1 = h1 + EP1(e1) + CH(e1, f1, g1) + d_k_90r[round] + m_round;
			uint32_t t2 = EP0(a1) + MAJ(a1, b1, c1);
			h1 = g1; g1 = f1; f1 = e1; e1 = d1 + t1; d1 = c1; c1 = b1; b1 = a1; a1 = t1 + t2;
		}

		// Process second input (bit flip)
		const uint8_t* data2 = input_data2 + block_idx * 64;
		__shared__ uint32_t shared_m2[90 * 32];
		uint32_t* m2 = shared_m2 + (threadIdx.x / 32) * 90;

		for (int word_idx = 0; word_idx < 16; ++word_idx) {
			int byte_offset = word_idx * 4;
			uint32_t word = (data2[byte_offset] << 24) | (data2[byte_offset + 1] << 16) |
						   (data2[byte_offset + 2] << 8) | data2[byte_offset + 3);
			uint32_t mask = (lane_id == word_idx) ? 0xFFFFFFFF : 0;
			m2[word_idx] = (word & mask) | (m2[word_idx] & ~mask);
		}
		__syncwarp();

		for (int i = 16; i < 90; ++i) {
			uint32_t m_i_minus_16 = m2[i - 16];
			uint32_t m_i_minus_15 = m2[i - 15];
			uint32_t m_i_minus_7 = m2[i - 7];
			uint32_t m_i_minus_2 = m2[i - 2];
			uint32_t new_word = SIG1(m_i_minus_2) + m_i_minus_7 + SIG0(m_i_minus_15) + m_i_minus_16;
			uint32_t mask = (lane_id == (i % 32)) ? 0xFFFFFFFF : 0;
			m2[i] = (new_word & mask) | (m2[i] & ~mask);
		}
		__syncwarp();

		uint32_t a2 = 0, b2 = 0, c2 = 0, d2 = 0, e2 = 0, f2 = 0, g2 = 0, h2 = 0;

		for (int state_idx = 0; state_idx < 8; ++state_idx) {
			uint32_t state_val = initial_states[state_idx];
			uint32_t mask = (lane_id == state_idx) ? 0xFFFFFFFF : 0;
			a2 = (state_val & mask) | (a2 & ~mask);
			b2 = (state_val & mask) | (b2 & ~mask);
			c2 = (state_val & mask) | (c2 & ~mask);
			d2 = (state_val & mask) | (d2 & ~mask);
			e2 = (state_val & mask) | (e2 & ~mask);
			f2 = (state_val & mask) | (f2 & ~mask);
			g2 = (state_val & mask) | (g2 & ~mask);
			h2 = (state_val & mask) | (h2 & ~mask);
		}

		for (int round = 0; round < 90; ++round) {
			uint32_t m_round = m2[round];
			uint32_t t1 = h2 + EP1(e2) + CH(e2, f2, g2) + d_k_90r[round] + m_round;
			uint32_t t2 = EP0(a2) + MAJ(a2, b2, c2);
			h2 = g2; g2 = f2; f2 = e2; e2 = d2 + t1; d2 = c2; c2 = b2; b2 = a2; a2 = t1 + t2;
		}

		uint64_t end_time = clock64();

		// Store timing result (only thread 0 per warp)
		if (lane_id == 0 && timing_results) {
			timing_results[block_idx] = end_time - start_time;
		}

		// Store results for verification
		if (lane_id < 8) {
			uint32_t* out_state1 = output_states + block_idx * 16;
			uint32_t* out_state2 = output_states + block_idx * 16 + 8;
			uint32_t state_values1[8] = {a1, b1, c1, d1, e1, f1, g1, h1};
			uint32_t state_values2[8] = {a2, b2, c2, d2, e2, f2, g2, h2};
			out_state1[lane_id] = initial_states[lane_id] + state_values1[lane_id];
			out_state2[lane_id] = initial_states[lane_id] + state_values2[lane_id];
		}
	}
}

// Host function for CUDA constant-time regression testing
extern "C" hipError_t launch_sha256_90r_cuda_timing_test(
	const uint8_t* input_data1,
	const uint8_t* input_data2,
	uint32_t* output_states,
	size_t num_blocks,
	uint64_t* timing_results,
	int threads_per_block = 256
) {
	hipError_t cudaStatus;

	uint8_t* d_input1 = nullptr;
	uint8_t* d_input2 = nullptr;
	uint32_t* d_output = nullptr;
	uint64_t* d_timing = nullptr;

	size_t input_size = num_blocks * 64 * sizeof(uint8_t);
	size_t output_size = num_blocks * 16 * sizeof(uint32_t); // 2 hashes per block
	size_t timing_size = num_blocks * sizeof(uint64_t);

	// Allocate device memory
	cudaStatus = hipMalloc(&d_input1, input_size);
	if (cudaStatus != hipSuccess) return cudaStatus;

	cudaStatus = hipMalloc(&d_input2, input_size);
	if (cudaStatus != hipSuccess) {
		hipFree(d_input1);
		return cudaStatus;
	}

	cudaStatus = hipMalloc(&d_output, output_size);
	if (cudaStatus != hipSuccess) {
		hipFree(d_input1);
		hipFree(d_input2);
		return cudaStatus;
	}

	cudaStatus = hipMalloc(&d_timing, timing_size);
	if (cudaStatus != hipSuccess) {
		hipFree(d_input1);
		hipFree(d_input2);
		hipFree(d_output);
		return cudaStatus;
	}

	// Copy input data
	cudaStatus = hipMemcpy(d_input1, input_data1, input_size, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) goto cleanup;

	cudaStatus = hipMemcpy(d_input2, input_data2, input_size, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) goto cleanup;

	// Copy constants
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(d_k_90r), k_90r, sizeof(k_90r));
	if (cudaStatus != hipSuccess) goto cleanup;

	// Calculate grid dimensions
	int warps_per_block = threads_per_block / 32;
	int blocks_per_grid = (num_blocks + warps_per_block - 1) / warps_per_block;

	// Launch timing test kernel
	sha256_90r_constant_time_test_kernel<<<blocks_per_grid, threads_per_block>>>(
		d_input1, d_input2, d_output, num_blocks, d_timing
	);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) goto cleanup;

	// Copy results back
	cudaStatus = hipMemcpy(output_states, d_output, output_size, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) goto cleanup;

	if (timing_results) {
		cudaStatus = hipMemcpy(timing_results, d_timing, timing_size, hipMemcpyDeviceToHost);
	}

cleanup:
	hipFree(d_input1);
	hipFree(d_input2);
	hipFree(d_output);
	hipFree(d_timing);

	return cudaStatus;
}

// Performance comparison: original vs warp-optimized kernel
extern "C" hipError_t launch_sha256_90r_cuda_batch_original(
	const uint8_t* input_data,
	uint32_t* output_states,
	size_t num_blocks,
	int threads_per_block = 256
) {
	hipError_t cudaStatus;

	// Allocate device memory
	uint8_t* d_input = nullptr;
	uint32_t* d_output = nullptr;

	size_t input_size = num_blocks * 64 * sizeof(uint8_t);
	size_t output_size = num_blocks * 8 * sizeof(uint32_t);

	cudaStatus = hipMalloc(&d_input, input_size);
	if (cudaStatus != hipSuccess) return cudaStatus;

	cudaStatus = hipMalloc(&d_output, output_size);
	if (cudaStatus != hipSuccess) {
		hipFree(d_input);
		return cudaStatus;
	}

	// Copy input data to device
	cudaStatus = hipMemcpy(d_input, input_data, input_size, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		hipFree(d_input);
		hipFree(d_output);
		return cudaStatus;
	}

	// Calculate grid dimensions
	int blocks_per_grid = (num_blocks + threads_per_block - 1) / threads_per_block;

	// Prefetch constants to GPU
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(d_k_90r), k_90r, sizeof(k_90r));
	if (cudaStatus != hipSuccess) {
		hipFree(d_input);
		hipFree(d_output);
		return cudaStatus;
	}

	// Launch original kernel
	sha256_90r_batch_kernel<<<blocks_per_grid, threads_per_block>>>(
		d_input, d_output, num_blocks
	);

	// Check for kernel launch errors
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		hipFree(d_input);
		hipFree(d_output);
		return cudaStatus;
	}

	// Copy results back to host
	cudaStatus = hipMemcpy(output_states, d_output, output_size, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		hipFree(d_input);
		hipFree(d_output);
		return cudaStatus;
	}

	// Cleanup
	hipFree(d_input);
	hipFree(d_output);

	return hipSuccess;
}
